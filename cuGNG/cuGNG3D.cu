#include "hip/hip_runtime.h"
/********************************************************************
*  cuGNG3D.cu
*  CUDA implementation of Growing Neural Gas Networkd for 3D input Data
*********************************************************************/

#define debug_msg 1

#include <stdio.h>
#include <vector>
#include <stdlib.h>
#include <cutil_inline.h>
#include <float.h>

#include "sharedmem.cuh"
#include "reduction.h"
#include "Neurona.h"
#include "cuGNG3D.h"

using namespace std;

enum ReduceType
{
    REDUCE_INT,
    REDUCE_FLOAT,
    REDUCE_DOUBLE
};

/* retorna "a - b" en segundos */
double performancecounter_diff(LARGE_INTEGER *a, LARGE_INTEGER *b)
{
  LARGE_INTEGER freq;
  QueryPerformanceFrequency(&freq);
  return (double)(a->QuadPart - b->QuadPart) / (double)freq.QuadPart;
}

void checkCUDAError(const char* msg);

/************************************************************************/
/* Init CUDA                                                            */
/************************************************************************/
bool init_cuda_device(int device_number)
{
	int count = 0;
	int i = 0;

	hipGetDeviceCount(&count);
	if(count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	hipDeviceProp_t prop;
	for(i = 0; i < count; i++) {
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if(prop.major >= 1) {
				break;
			}
		}
	}
	if(i == count) {
		fprintf(stderr, "There is no device supporting CUDA.\n");
		return false;
	}
	hipSetDevice(device_number);

	hipGetDeviceProperties(&prop, 1);
	printf("CUDA device: %s initialized.\n",prop.name);

	return true;
}

bool isPow2(unsigned int x)
{
    return ((x&(x-1))==0);
}

unsigned int nextPow2( unsigned int x ) {
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}

////////////////////////////////////////////////////////////////////////////////
// Compute the number of threads and blocks to use for the given reduction kernel
// For the kernels >= 3, we set threads / block to the minimum of maxThreads and
// n/2. For kernels < 3, we set to the minimum of maxThreads and n.  For kernel 
// 6, we observe the maximum specified number of blocks, because each thread in 
// that kernel can process a variable number of elements.
////////////////////////////////////////////////////////////////////////////////
void getNumBlocksAndThreads(int n,int &blocks, int &threads)
{     
	int maxBlocks=64;
	int maxThreads=256;

    threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
	blocks = (n + (threads * 2 - 1)) / (threads * 2);
}

// Constantes
__constant__ float  COEF_VECINAS,COEF_GANADORA;
__constant__ int  size;

//FUNCION REDUCE MIN NEURONA 3
__global__ void
reduceMinNeurona3_Min2(Neurona *g_idata, reduceSTRUCT2 *g_odata,float x,float y,float z)
{
    SharedMemory<reduceSTRUCT2> smem;
    reduceSTRUCT2 *sdata = smem.getPointer();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

	if( i<size && g_idata[i].utilizada )
	{
		sdata[tid].distancia1 = (g_idata[i].pesos[0]-x)*(g_idata[i].pesos[0]-x)+(g_idata[i].pesos[1]-y)*(g_idata[i].pesos[1]-y)+(g_idata[i].pesos[2]-z)*(g_idata[i].pesos[2]-z);
		sdata[tid].indice1 = i;
		sdata[tid].distancia2 = 99999999;
		sdata[tid].indice2 = -1;
	}
	else
	{
		sdata[tid].distancia1 = 99999999;
		sdata[tid].indice1 = -1;
		sdata[tid].distancia2 = 99999999;
		sdata[tid].indice2 = -1;
	}

	if (i + blockDim.x < size && g_idata[i+blockDim.x].utilizada )
	{
		float distancia = (g_idata[i+blockDim.x].pesos[0]-x)*(g_idata[i+blockDim.x].pesos[0]-x)+(g_idata[i+blockDim.x].pesos[1]-y)*(g_idata[i+blockDim.x].pesos[1]-y)+(g_idata[i+blockDim.x].pesos[2]-z)*(g_idata[i+blockDim.x].pesos[2]-z);
	
		if(sdata[tid].distancia1 > distancia) 
		{
			sdata[tid].distancia2 = sdata[tid].distancia1; 
			sdata[tid].indice2 = sdata[tid].indice1;
			sdata[tid].distancia1 = distancia; 
			sdata[tid].indice1 = i+blockDim.x;
		}
		else
		{
			sdata[tid].distancia2 = distancia;
			sdata[tid].indice2 = i+blockDim.x;
		}
	}

    __syncthreads();

    // do reduction in shared mem
    for(unsigned int s=blockDim.x/2; s>0; s>>=1) 
    {
        if (tid < s) 
        {
			if(sdata[tid].distancia1 > sdata[tid + s].distancia1)
			{
				if( sdata[tid].distancia1 < sdata[tid + s].distancia2 )
				{
					sdata[tid].distancia2 = sdata[tid].distancia1; 
					sdata[tid].indice2 = sdata[tid].indice1;
				}
				else
				{
					sdata[tid].distancia2 = sdata[tid + s].distancia2; 
					sdata[tid].indice2 = sdata[tid + s].indice2;
				}
				sdata[tid].distancia1 = sdata[tid + s].distancia1;
				sdata[tid].indice1 = sdata[tid + s].indice1;
			}
			else
			{
				if(sdata[tid].distancia2 > sdata[tid + s].distancia1)
				{
					sdata[tid].distancia2 = sdata[tid + s].distancia1;
					sdata[tid].indice2 = sdata[tid + s].indice1;
				}
			}
        }
        __syncthreads();
    }

    // write result for this block to global mem 
    if (tid == 0)
	{	
		g_odata[blockIdx.x].distancia1 = sdata[0].distancia1;
		g_odata[blockIdx.x].indice1 = sdata[0].indice1;
		g_odata[blockIdx.x].distancia2 = sdata[0].distancia2;
		g_odata[blockIdx.x].indice2 = sdata[0].indice2;
	}
}

__global__ void
adjustWeightsWinningNeurons(auxVECINA *mat,Neurona *g_idata, reduceSTRUCT2 *g_odata, unsigned int numBlocks,float x,float y,float z)
{

	float minimo,minimo2;
	int iminimo,iminimo2;

	minimo=g_odata[0].distancia1;
	iminimo=g_odata[0].indice1;
	minimo2=g_odata[0].distancia2;
	iminimo2=g_odata[0].indice2;

	//printf("minimo1 %f iminimo1 %d",minimo,iminimo);
	//printf("minimo2 %f iminimo2 %d",minimo2,iminimo2);

	//Obtenemos los m�nimos finales en gpu
	for(int i=1;i<numBlocks;i++)
	{
		if( g_odata[i].distancia1 < minimo )
		{
			if( g_odata[i].distancia2 < minimo )
			{
				minimo2 = g_odata[i].distancia2;
				iminimo2 = g_odata[i].indice2;
			}
			else
			{
				minimo2 = minimo;
				iminimo2 = iminimo;
			}
			minimo=g_odata[i].distancia1;
			iminimo=g_odata[i].indice1;
		}
		else
		{
			if( g_odata[i].distancia1 < minimo2 )
			{
				minimo2 = g_odata[i].distancia1;
				iminimo2 = g_odata[i].indice1;
			}
		}
	}

	for(int i=1;i<=mat[iminimo*(100+1)].indice_vecina;i++)
	{
		mat[iminimo*(100+1)+i].antiguedad = mat[iminimo*(100+1)+i].antiguedad + 1;
	}

	// Se incrementan los errores de la neurona ganadora
	g_idata[iminimo].error = g_idata[iminimo].error + minimo;
	g_idata[iminimo].error_parcial = g_idata[iminimo].error_parcial + minimo;

	g_idata[iminimo].pesos[0] += COEF_GANADORA*((float)x-g_idata[iminimo].pesos[0]);
	g_idata[iminimo].pesos[1] += COEF_GANADORA*((float)y-g_idata[iminimo].pesos[1]);
	g_idata[iminimo].pesos[2] += COEF_GANADORA*((float)z-g_idata[iminimo].pesos[2]);

	// Se modifican los pesos de las neuronas vecinas
	for(int i=1;i<=mat[iminimo*(100+1)].indice_vecina;i++)
	{
		g_idata[mat[iminimo*(100+1)+i].indice_vecina].pesos[0]+=COEF_VECINAS*((float)x-g_idata[mat[iminimo*(100+1)+i].indice_vecina].pesos[0]);
		g_idata[mat[iminimo*(100+1)+i].indice_vecina].pesos[1]+=COEF_VECINAS*((float)y-g_idata[mat[iminimo*(100+1)+i].indice_vecina].pesos[1]);
		g_idata[mat[iminimo*(100+1)+i].indice_vecina].pesos[2]+=COEF_VECINAS*((float)z-g_idata[mat[iminimo*(100+1)+i].indice_vecina].pesos[2]);
	}

	bool encontrado=false;

	for(int i=1;i<=mat[iminimo*(100+1)].indice_vecina && encontrado == false;i++)
	{
		if( mat[iminimo*(100+1)+i].indice_vecina == iminimo2)
		{
			encontrado=true;
			mat[iminimo*(100+1)+i].antiguedad = 0;
		}
	}

	if(encontrado==false)
	{
		int num = mat[iminimo*(100+1)].indice_vecina;
		mat[iminimo*(100+1)+num+1].antiguedad = 0;
		mat[iminimo*(100+1)+num+1].indice_vecina = iminimo2;
		mat[iminimo*(100+1)].indice_vecina = mat[iminimo*(100+1)].indice_vecina + 1;
	}

	encontrado=false;
	for(int i=1;i<=mat[iminimo2*(100+1)].indice_vecina && encontrado == false;i++)
	{
		if( mat[iminimo2*(100+1)+i].indice_vecina == iminimo )
		{
			encontrado=true;
			mat[iminimo2*(100+1)+i].antiguedad = 0;
		}
	}

	if(encontrado==false)
	{
		int num = mat[iminimo2*(100+1)].indice_vecina;
		mat[iminimo2*(100+1)+num+1].antiguedad = 0;
		mat[iminimo2*(100+1)+num+1].indice_vecina = iminimo;
		mat[iminimo2*(100+1)].indice_vecina = mat[iminimo2*(100+1)].indice_vecina + 1; 
	}
}


int init_mem_cuda_gng(Neurona* &d_idata, reduceSTRUCT2* &d_odata,auxVECINA* &d_matVecinas,int max_neurons)
{
	int maxThreads = 256;  // number of threads per block
	int maxBlocks = 64;

	// allocate mem for the result on host side
	//reduceSTRUCT2* h_odata = (reduceSTRUCT2*) malloc(numBlocks*sizeof(reduceSTRUCT2));

	// size of bytes that gng need to store neurons
	unsigned int bytes = max_neurons * sizeof(Neurona);

	cutilSafeCallNoSync( hipMalloc((void**) &d_idata, bytes) );
	cutilSafeCallNoSync( hipMalloc((void**) &d_odata, maxBlocks*sizeof(reduceSTRUCT2)) );
	cutilSafeCallNoSync( hipMalloc((void**) &d_matVecinas, ((max_neurons*100)+max_neurons)*sizeof(auxVECINA)) );

	//printf(" bytes d_idata = %i \n",bytes);
	//printf(" bytes d_odata = %i \n",maxBlocks*sizeof(reduceSTRUCT2));
	//printf(" d_matVecinas = %i \n",((max_neurons*100)+max_neurons)*sizeof(auxVECINA));

	return 0;
}

int destroy_mem_cuda_gng(Neurona* d_idata, reduceSTRUCT2* d_odata )
{
  cutilSafeCallNoSync(hipFree(d_idata));
  cutilSafeCallNoSync(hipFree(d_odata));
  return 0;
}

/************************************************************************/
/* CU_AjusteGNG                                                         */
/************************************************************************/
int CU_AjusteGNG(Neurona *h_idata,Neurona *d_idata,reduceSTRUCT2* d_odata,auxVECINA* h_matVecinas,auxVECINA* d_matVecinas,int numThreads,int numBlocks,int size,int NUM_ENTRADAS,float *nube_puntos,int NUM_PUNTOS)
{
	int entrada;
	float minimo,minimo2;
	int iminimo=0,iminimo2=1;
	float error;
	
	float COEF_GANADORA=(float)0.1;
    float COEF_VECINAS=(float)0.01;
	int UMBRAL_DATOS=127;

    dim3 dimBlock(numThreads, 1, 1);
    dim3 dimGrid(numBlocks, 1, 1);
    int smemSize = numThreads * sizeof(Neurona);

	int al;
	float x,y,z;

	error=0;
	minimo=minimo2=FLT_MAX;

	////Reservamos memoria para el vector neuronas que vamos a copiar a la memoria global GPU
    unsigned int bytes = size * sizeof(Neurona);

	/*hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start,0);*/
    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
	/*hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf("Time to transfer host -> device: %3.5f ms -- bytes= %d\n",elapsedTime,bytes);
	hipEventDestroy(start);
	hipEventDestroy(stop);*/

	/****************************************************************************************/
	//Iteramos para las N_ENTRADAS modificando los pesos de la neurona ganadora y sus vecinas
	/****************************************************************************************/

	hipMemcpyToSymbol(HIP_SYMBOL("COEF_GANADORA"),    &COEF_GANADORA,    sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL("COEF_VECINAS"),    &COEF_VECINAS,    sizeof(float));

	int aux;
	if( isPow2(size) )
	{
		hipMemcpyToSymbol(HIP_SYMBOL("size"),    &size,    sizeof(size));
	}else
	{
		aux = nextPow2(size);
		hipMemcpyToSymbol(HIP_SYMBOL("size"),    &aux,    sizeof(aux));	
	}

	// **************************************************************************************************
	// *************************** COPIAMOS EL std::vector Y REALIZAMOS UNA COPIA ***********************
	// **************************************************************************************************
	
	for(int i=0;i<size;i++)
	{
		if( h_idata[i].utilizada == false )
		{
			h_matVecinas[i*(100+1)].indice_vecina = -1;
			h_matVecinas[i*(100+1)].antiguedad = -1;
		}
		else
		{	
			h_matVecinas[i*(100+1)].indice_vecina = h_idata[i].vecinas.size();

			for(int j=0;j<h_idata[i].vecinas.size();j++)
			{
				h_matVecinas[i*(100+1)+j+1].indice_vecina = h_idata[i].vecinas[j].vecina;
				h_matVecinas[i*(100+1)+j+1].antiguedad = h_idata[i].vecinas[j].antiguedad;
			}
		}
	}

	hipMemcpy(d_matVecinas, h_matVecinas,((size*100)+size)*sizeof(auxVECINA), hipMemcpyHostToDevice);

	// **************************************************************************************************
	// **************************************************************************************************
	// **************************************************************************************************

	for (entrada=0;entrada<NUM_ENTRADAS;entrada++)
	{
		al=rand()%(NUM_PUNTOS);

		x=nube_puntos[al*3];
		y=nube_puntos[al*3+1];
		z=nube_puntos[al*3+2];

		reduceMinNeurona3_Min2<<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, x, y, z);

		adjustWeightsWinningNeurons<<<1,1>>>(d_matVecinas, d_idata, d_odata, numBlocks,x,y, z);
	}

	//Copiamos la matriz despu�s de las lambda entradas y actualizamos el vector de neuronas host
	hipMemcpy(h_matVecinas, d_matVecinas, sizeof(auxVECINA)*((size*100)+size), hipMemcpyDeviceToHost);
	
	//Copiamos los cambios realizados en el vector de neuronas device
	cutilSafeCall( hipMemcpy(h_idata, d_idata, bytes, hipMemcpyDeviceToHost) );

	for(int i=0;i<size;i++)
	{
		int numVecinas = h_matVecinas[i*(100+1)].indice_vecina;
		
		//Borramos las vecinas de esa neurona y actualizamos
		if(numVecinas > 0)	h_idata[i].vecinas.clear();

		for(int j=1;j<=numVecinas;j++)
		{
			VECINAS vec;
			vec.vecina = h_matVecinas[i*(100+1)+j].indice_vecina;
			vec.antiguedad = h_matVecinas[i*(100+1)+j].antiguedad;
			h_idata[i].vecinas.push_back(vec);
		}
	}
	
	return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}
